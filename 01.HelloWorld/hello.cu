#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
	printf("Hello World from GPU!\n");
}

int main() {
	printf("Hello World!\n");
	cuda_hello<<<1,1>>>();

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
				hipGetErrorString(cudaerr));
	return 0;
}
